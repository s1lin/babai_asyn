#include "hip/hip_runtime.h"
#include "../babai_asyn_c++/src/source/cils.cpp"
#include "cils_cuda_solvers.cuh"
#include "hip/hip_runtime_api.h"
#include <ctime>

namespace cils {

    template<typename scalar, typename index, index n>
    void testDevice(index devID) {
        // Check if we can run. Maybe do something more...
        hipDeviceProp_t deviceProp{};
        hipGetDeviceProperties(&deviceProp, devID);
        if (deviceProp.major == 9999 && deviceProp.minor == 9999) {   /* Simulated device. */
            printf("There is no device supporting CUDA.\n");
        } else {
            index cores = 0;
            index mp = deviceProp.multiProcessorCount;
            switch (deviceProp.major){
                case 2: // Fermi
                    if (deviceProp.minor == 1) cores = mp * 48;
                    else cores = mp * 32;
                    break;
                case 3: // Kepler
                    cores = mp * 192;
                    break;
                case 5: // Maxwell
                    cores = mp * 128;
                    break;
                case 6: // Pascal
                    if ((deviceProp.minor == 1) || (deviceProp.minor == 2)) cores = mp * 128;
                    else if (deviceProp.minor == 0) cores = mp * 64;
                    else printf("Unknown device type\n");
                    break;
                case 7: // Volta and Turing
                    if ((deviceProp.minor == 0) || (deviceProp.minor == 5)) cores = mp * 64;
                    else printf("Unknown device type\n");
                    break;
                case 8: // Ampere
                    if (deviceProp.minor == 0) cores = mp * 64;
                    else if (deviceProp.minor == 6) cores = mp * 128;
                    else printf("Unknown device type\n");
                    break;
                default:
                    printf("Unknown device type\n");
                    break;
            }
            printf("Using GPU device number %d, with %d CUDA cores.\n", devID, cores);
        }
    }

    template<typename scalar, typename index, bool is_read, index n>
    returnType<scalar, index>
    cils<scalar, index, is_read, n>::cils_babai_search_cuda(index nswp, vector<index> *z_B) {
        scalar *z_B_d, *z_B_h, *y_A_d, *R_A_d;

        z_B_h = (scalar *) malloc(n * sizeof(scalar));

        hipMallocManaged(&z_B_d, n * sizeof(scalar));
        hipMallocManaged(&y_A_d, n * sizeof(scalar));
        hipMallocManaged(&R_A_d, R_A->size * sizeof(scalar));

        index end_1 = n - 1;
        z_B->at(end_1) = round(y_A->x[end_1] / R_A->x[(n * end_1) + end_1 - ((end_1 * (end_1 + 1)) / 2)]);

        for (index row = 0; row < n; row++) {
            z_B_h[row] = z_B->at(row);
        }

        hipMemcpy(y_A_d, y_A->x, n * sizeof(scalar), hipMemcpyHostToDevice);
        hipMemcpy(R_A_d, R_A->x, R_A->size * sizeof(scalar), hipMemcpyHostToDevice);
        hipMemcpy(z_B_d, z_B_h, n * sizeof(scalar), hipMemcpyHostToDevice);

        index tileSize = 4;
        // Optimized kernel
        index nTiles = n / tileSize + (n % tileSize == 0 ? 0 : 1);
        index gridHeight = n / tileSize + (n % tileSize == 0 ? 0 : 1);
        index gridWidth = n / tileSize + (n % tileSize == 0 ? 0 : 1);
        dim3 dGrid(gridHeight, gridWidth), dBlock(tileSize, tileSize);

        std::clock_t start = std::clock();
        for (index k = 0; k < nswp; k++) {
            cuda::babai_solve_cuda<scalar, index, n><<<nTiles, tileSize>>>(R_A_d, y_A_d, z_B_d);
        }

        hipDeviceSynchronize();
        scalar run_time = (std::clock() - start) / (scalar) CLOCKS_PER_SEC;
        hipMemcpy(z_B_h, z_B_d, n * sizeof(scalar), hipMemcpyDeviceToHost);
        for (index row = 0; row < n; row++) {
            z_B->at(row) = z_B_h[row];
        }

        hipFree(z_B_d);
        hipFree(y_A_d);
        hipFree(R_A_d);
        free(z_B_h);

        returnType<scalar, index> reT = {*z_B, run_time, 0, 0};
        return reT;
    }

    template<typename scalar, typename index, bool is_read, index n>
    returnType<scalar, index>
    cils<scalar, index, is_read, n>::cils_block_search_cuda(index nswp, scalar stop, vector<index> *z_B, vector<index> *d) {

        index ds = d->size(), dx = d->at(ds - 1);
        if (ds == 1) {
            if (d->at(0) == 1) {
                z_B->at(0) = round(y_A->x[0] / R_A->x[0]);
                return {*z_B, 0, 0, 0};
            } else {
                vector<scalar> R_B = find_block_Rii(R_A, 0, n, 0, n, n);
                vector<scalar> y_B = find_block_x(y_A, 0, n);
                return {ils_search(&R_B, &y_B), 0, 0, 0};
            }
        } else if (ds == n) {
            //Find the Babai point by OpenMP
            return cils_babai_search_cuda(nswp, z_B);
        }

        scalar *z_B_d, *z_B_h, *y_A_d, *R_A_d;
        index *d_A_d, *d_A_h;

        z_B_h = (scalar *) malloc(n * sizeof(scalar));
        d_A_h = (index *) malloc(ds * sizeof(index));

        hipMallocManaged(&z_B_d, n * sizeof(scalar));
        hipMallocManaged(&y_A_d, n * sizeof(scalar));
        hipMallocManaged(&R_A_d, R_A->size * sizeof(scalar));
        hipMallocManaged(&d_A_d, ds * sizeof(index));

        for (index row = 0; row < ds; row++) {
            z_B_h[row] = z_B->at(row);
            d_A_h[row] = d->at(row);
        }
        for (index row = ds; row < n; row++){
            z_B_h[row] = z_B->at(row);
        }

        hipMemcpy(y_A_d, y_A->x, n * sizeof(scalar), hipMemcpyHostToDevice);
        hipMemcpy(R_A_d, R_A->x, R_A->size * sizeof(scalar), hipMemcpyHostToDevice);
        hipMemcpy(z_B_d, z_B_h, n * sizeof(scalar), hipMemcpyHostToDevice);
        hipMemcpy(d_A_d, d_A_h, n * sizeof(scalar), hipMemcpyHostToDevice);

        // Optimized kernel
        index nTiles = n / ds + (n % ds == 0 ? 0 : 1);
        cout<<nTiles;
        index gridHeight = n / ds + (n % ds == 0 ? 0 : 1);
        index gridWidth = n / ds + (n % ds == 0 ? 0 : 1);
        dim3 dGrid(gridHeight, gridWidth), dBlock(ds, ds);

        std::clock_t start = std::clock();
        for (index k = 0; k < nswp; k++) {
            cuda::block_solve_cuda<scalar, index, n><<<1, ds>>>(R_A_d, y_A_d, d_A_d, ds, z_B_d);
        }

        hipDeviceSynchronize();
        scalar run_time = (std::clock() - start) / (scalar) CLOCKS_PER_SEC;
        hipMemcpy(z_B_h, z_B_d, n * sizeof(scalar), hipMemcpyDeviceToHost);
        for (index row = 0; row < n; row++) {
            z_B->at(row) = z_B_h[row];
        }

        hipFree(z_B_d);
        hipFree(y_A_d);
        hipFree(R_A_d);
        hipFree(d_A_d);
        free(z_B_h);

        returnType<scalar, index> reT = {*z_B, run_time, 0, 0};
        return reT;
    }
}
