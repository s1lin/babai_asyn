#include "hip/hip_runtime.h"
#include "../babai_asyn_c++/src/source/cils.cpp"
#include "cils_cuda_solvers.cuh"
#include <ctime>

namespace cils {

    template<typename scalar, typename index, index n>
    void testDevice(index devID) {
        // Check if we can run. Maybe do something more...
        hipDeviceProp_t deviceProp{};
        hipGetDeviceProperties(&deviceProp, devID);
        if (deviceProp.major == 9999 && deviceProp.minor == 9999) {   /* Simulated device. */
            printf("There is no device supporting CUDA.\n");
            hipDeviceSynchronize();
        } else
            printf("Using GPU device number %d.\n", devID);
    }

    template<typename scalar, typename index, bool is_read, index n>
    returnType<scalar, index>
    cils<scalar, index, is_read, n>::cils_babai_search_cuda(index nswp, vector<index> *z_B) {
        scalar *z_B_d, *z_B_h, *y_A_c, *R_A_c;

        z_B_h = (scalar *) malloc(n * sizeof(scalar));

        hipMallocManaged(&z_B_d, n * sizeof(scalar));
        hipMallocManaged(&y_A_c, n * sizeof(scalar));
        hipMallocManaged(&R_A_c, R_A->size * sizeof(scalar));

        index end_1 = n - 1;
        z_B->at(end_1) = round(y_A->x[end_1] / R_A->x[(n * end_1) + end_1 - ((end_1 * (end_1 + 1)) / 2)]);

        for (index row = 0; row < n; row++) {
            z_B_h[row] = z_B->at(row);
        }

        hipMemcpy(y_A_c, y_A->x, n * sizeof(scalar), hipMemcpyHostToDevice);
        hipMemcpy(R_A_c, R_A->x, R_A->size * sizeof(scalar), hipMemcpyHostToDevice);
        hipMemcpy(z_B_d, z_B_h, n * sizeof(scalar), hipMemcpyHostToDevice);

        index tileSize = 256;
        // Optimized kernel
        index nTiles = n / tileSize + (n % tileSize == 0 ? 0 : 1);
        index gridHeight = n / tileSize + (n % tileSize == 0 ? 0 : 1);
        index gridWidth = n / tileSize + (n % tileSize == 0 ? 0 : 1);
        dim3 dGrid(gridHeight, gridWidth), dBlock(tileSize, tileSize);

        std::clock_t start = std::clock();
        for (index k = 0; k < nswp; k++) {
            babai_solve_cuda<scalar, index, n><<<nTiles, tileSize>>>(R_A_c, y_A_c, z_B_d);
        }

        hipDeviceSynchronize();
        scalar run_time = (std::clock() - start) / (scalar) CLOCKS_PER_SEC;
        hipMemcpy(z_B_h, z_B_d, n * sizeof(scalar), hipMemcpyDeviceToHost);
        for (index row = 0; row < n; row++) {
            z_B->at(row) = z_B_h[row];
        }

        hipFree(z_B_d);
        hipFree(y_A_c);
        hipFree(R_A_c);
        free(z_B_h);

        returnType<scalar, index> reT = {*z_B, run_time, 0, 0};
        return reT;
    }

}
