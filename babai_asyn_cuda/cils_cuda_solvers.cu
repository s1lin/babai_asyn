#include "hip/hip_runtime.h"
#include "../babai_asyn_c++/src/source/cils.cpp"
#include "cils_cuda_solvers.cuh"
#include <ctime>

namespace cils {

    template<typename scalar, typename index, index n>
    void testDevice(index devID) {
        // Check if we can run. Maybe do something more...
        hipDeviceProp_t deviceProp{};
        hipGetDeviceProperties(&deviceProp, devID);
        if (deviceProp.major == 9999 && deviceProp.minor == 9999) {   /* Simulated device. */
            printf("There is no device supporting CUDA.\n");
            hipDeviceSynchronize();
        } else
            printf("Using GPU device number %d.\n", devID);
    }

    template<typename scalar, typename index, bool is_read, index n>
    returnType<scalar, index>
    cils<scalar, index, is_read, n>::cils_babai_search_cuda(index nswp, vector<index> *z_B) {
        scalar *z_B_d, *z_B_h, *y_A_c, *R_A_c;

        z_B_h = (scalar *) malloc(n * sizeof(scalar));

        hipMallocManaged(&z_B_d, n * sizeof(scalar));
        hipMallocManaged(&y_A_c, n * sizeof(scalar));
        hipMallocManaged(&R_A_c, R_A->size * sizeof(scalar));

        index end_1 = n - 1;
        z_B->at(end_1) = round(y_A->x[end_1] / R_A->x[(n * end_1) + end_1 - ((end_1 * (end_1 + 1)) / 2)]);

        for (index row = 0; row < n; row++) {
            z_B_h[row] = z_B->at(row);
        }

        hipMemcpy(y_A_c, y_A->x, n * sizeof(scalar), hipMemcpyHostToDevice);
        hipMemcpy(R_A_c, R_A->x, R_A->size * sizeof(scalar), hipMemcpyHostToDevice);
        hipMemcpy(z_B_d, z_B_h, n * sizeof(scalar), hipMemcpyHostToDevice);

        index tileSize = 4;
        // Optimized kernel
        index nTiles = n / tileSize + (n % tileSize == 0 ? 0 : 1);
        index gridHeight = n / tileSize + (n % tileSize == 0 ? 0 : 1);
        index gridWidth = n / tileSize + (n % tileSize == 0 ? 0 : 1);
        dim3 dGrid(gridHeight, gridWidth), dBlock(tileSize, tileSize);

        std::clock_t start = std::clock();
        for (index k = 0; k < nswp; k++) {
            babai_solve_cuda<scalar, index, n><<<nTiles, tileSize>>>(R_A_c, y_A_c, z_B_d);
        }

        hipDeviceSynchronize();
        scalar run_time = (std::clock() - start) / (scalar) CLOCKS_PER_SEC;
        hipMemcpy(z_B_h, z_B_d, n * sizeof(scalar), hipMemcpyDeviceToHost);
        for (index row = 0; row < n; row++) {
            z_B->at(row) = z_B_h[row];
        }

        hipFree(z_B_d);
        hipFree(y_A_c);
        hipFree(R_A_c);
        free(z_B_h);

        returnType<scalar, index> reT = {*z_B, run_time, 0, 0};
        return reT;
    }

    template<typename scalar, typename index, bool is_read, index n>
    returnType<scalar, index>
    cils<scalar, index, is_read, n>::cils_block_search_cuda(index nswp, scalar stop, vector<index> *z_B, vector<index> *d) {

        index ds = d->size(), dx = d->at(ds - 1);
        if (ds == 1) {
            if (d->at(0) == 1) {
                z_B->at(0) = round(y_A->x[0] / R_A->x[0]);
                return {*z_B, 0, 0, 0};
            } else {
                vector<scalar> R_B = find_block_Rii(R_A, 0, n, 0, n, n);
                vector<scalar> y_B = find_block_x(y_A, 0, n);
                return {ils_search(&R_B, &y_B), 0, 0, 0};
            }
        } else if (ds == n) {
            //Find the Babai point by OpenMP
            return cils_babai_search_cuda(nswp, nswp, z_B);
        }

        scalar *z_B_d, *z_B_h, *y_A_c, *R_A_c;

        z_B_h = (scalar *) malloc(n * sizeof(scalar));

        hipMallocManaged(&z_B_d, n * sizeof(scalar));
        hipMallocManaged(&y_A_c, n * sizeof(scalar));
        hipMallocManaged(&R_A_c, R_A->size * sizeof(scalar));

        for (index row = 0; row < n; row++) {
            z_B_h[row] = z_B->at(row);
        }

        hipMemcpy(y_A_c, y_A->x, n * sizeof(scalar), hipMemcpyHostToDevice);
        hipMemcpy(R_A_c, R_A->x, R_A->size * sizeof(scalar), hipMemcpyHostToDevice);
        hipMemcpy(z_B_d, z_B_h, n * sizeof(scalar), hipMemcpyHostToDevice);

        index tileSize = 4;
        // Optimized kernel
        index nTiles = n / tileSize + (n % tileSize == 0 ? 0 : 1);
        index gridHeight = n / tileSize + (n % tileSize == 0 ? 0 : 1);
        index gridWidth = n / tileSize + (n % tileSize == 0 ? 0 : 1);
        dim3 dGrid(gridHeight, gridWidth), dBlock(tileSize, tileSize);

        std::clock_t start = std::clock();
        for (index k = 0; k < nswp; k++) {
            block_solve_cuda<scalar, index, n><<<nTiles, tileSize>>>(R_A_c, y_A_c, z_B_d);
        }

        hipDeviceSynchronize();
        scalar run_time = (std::clock() - start) / (scalar) CLOCKS_PER_SEC;
        hipMemcpy(z_B_h, z_B_d, n * sizeof(scalar), hipMemcpyDeviceToHost);
        for (index row = 0; row < n; row++) {
            z_B->at(row) = z_B_h[row];
        }

        hipFree(z_B_d);
        hipFree(y_A_c);
        hipFree(R_A_c);
        free(z_B_h);

        returnType<scalar, index> reT = {*z_B, run_time, 0, 0};
        return reT;
    }
}
