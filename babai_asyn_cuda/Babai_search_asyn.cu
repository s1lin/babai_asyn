#include "hip/hip_runtime.h"
#include "Babai_search_asyn.h"
#include <ctime>

using Eigen::MatrixXd;
using Eigen::VectorXd;

__global__ void
find_raw_x0_cuda(int n, double *x_A, double *x_Next_A, const double *y_A, const double *R_sA) {

//    for (int j = 0; j < nswp; j++) {
//        double sum = 0;
//        for (int i = 1; i < n; i++) {
//            for (int col = n - i; col < n; col++) {
//                sum += R_sA[(n - 1 - i) * n - ((n - 1 - i) * (n - i)) / 2 + col] * raw_x_A[col];
//            }
//            raw_x_A[n - 1 - i] = round(
//                    (y_A[n - 1 - i] - sum) / R_sA[(n - 1 - i) * n - ((n - 1 - i) * (n - i)) / 2 + n - 1 - i]);
//            sum = 0;
//        }
//    }
//tiling
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        double sum = 0.0;

        // store index in register
        // Multiplication is not executed in every iteration.
        //int idx_Ai = idx * n;
        for (int col = idx + 1; col < n; col++)
            sum += R_sA[idx * n + col] * x_A[col];
        x_A[idx] = round((y_A[idx] - sum) / R_sA[idx * n + idx]);
    }

//int idx_Ai = idx * n;
//    for (int j=0; j<Nj; j++)
//        if (idx != j)
//            sum += R_sA[idx_Ai + j] * x_A[j];
}

void testDevice(int devID) {
    // Check if we can run. Maybe do something more...
    hipDeviceProp_t deviceProp;

    hipGetDeviceProperties(&deviceProp, devID);
    if (deviceProp.major == 9999 && deviceProp.minor == 9999) {   /* Simulated device. */
        printf("There is no device supporting CUDA.\n");
        hipDeviceReset();
    } else
        printf("Using GPU device number %d.\n", devID);
}

void run(int n, int nswp, Babai_search_asyn bsa){
    double *x, *x_A, *x_Next_A, *y_A, *R_sA;

    x = (double *) malloc(n * sizeof(double));

    hipMallocManaged(&x_A, n * sizeof(double));
    hipMallocManaged(&x_Next_A, n * sizeof(double));
    hipMallocManaged(&y_A, n * sizeof(double));
    hipMallocManaged(&R_sA, bsa.R_A.size() * sizeof(double));

    //x = bsa.x_A.data();
    x[n - 1] = round(bsa.y(n - 1) / bsa.R(n - 1, n - 1));
    for (int i = 0; i < n; i++) {
        y_A[i] = bsa.y_A[i];
    }
    for (int i = 0; i < bsa.R_A.size(); i++){
        R_sA[i] = bsa.R_A[i];
    }
    x_A[n - 1] = x_Next_A[n - 1] = x[n - 1];

    hipMemcpy(y_A, bsa.y_A.data(), n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(R_sA, bsa.R_A.data(),  bsa.R_A.size()*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(x_A, x, n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(x_Next_A, x, n * sizeof(double), hipMemcpyHostToDevice);

    int tileSize = 4;
    // Optimized kernel
    int nTiles = n / tileSize + (n % tileSize == 0 ? 0 : 1);
    int gridHeight = n / tileSize + (n % tileSize == 0 ? 0 : 1);
    int gridWidth = n / tileSize + (n % tileSize == 0 ? 0 : 1);
    //printf("w=%d, h=%d\n", gridWidth, gridHeight);
    dim3 dGrid(gridHeight, gridWidth), dBlock(tileSize, tileSize);

    std::clock_t start = std::clock();
    for (int k = 0; k < nswp; k++) {
        //if (k % 2)
        find_raw_x0_cuda<<<nTiles, tileSize>>>(n, x_A, x_Next_A, y_A, R_sA);
        //else
        //    find_raw_x0_cuda<<<nTiles, tileSize>>>(n, x_Next_A, x_A, y_A, R_sA);

    }
    hipDeviceSynchronize();
    double time = (std::clock() - start) / (double) CLOCKS_PER_SEC;

    double res = 0.0f;
    VectorXd x_result = VectorXd(n);

    for (int i = 0; i < n; i++) {
        x_result(i) = x_A[i];
    }

    res = (bsa.y - bsa.R * x_result).norm();

    printf("Sweep: %d, Res: %.5f, Run time: %fs\n", nswp, res, time);

    hipFree(x_A);
    hipFree(x_Next_A);
    hipFree(y_A);
    hipFree(R_sA);
    free(x);
}

int main() {

    testDevice(0);

    int n = 2048, n_jobs = 50;
    Babai_search_asyn bsa(n);

    bsa.init(true, 5);

    std::cout << "find_raw_x0" << std::endl;
    bsa.find_raw_x0();

    std::cout << "find_raw_x0_OMP" << std::endl;
    for(int nswp = 5; nswp <= n_jobs; nswp++)
        for(int j = 10; j <= n_jobs; j++)
            VectorXd x_par = bsa.find_raw_x0_OMP(j, nswp);

    for(int nswp = 10; nswp < 200; nswp += 10) {
        run(n, nswp, bsa);
    }

    return 0;

}
