#include "hip/hip_runtime.h"
#include "../babai_asyn_c++/src/include/SILS.h"
#include <ctime>

using Eigen::MatrixXd;
using Eigen::VectorXd;

namespace plt = matplotlibcpp;
__global__ void
find_raw_x0_cuda(int n, double *x_A, double *x_Next_A, const double *y_A, const double *R_sA) {

//    for (int j = 0; j < nswp; j++) {
//        double sum = 0;
//        for (int i = 1; i < n; i++) {
//            for (int col = n - i; col < n; col++) {
//                sum += R_sA[(n - 1 - i) * n - ((n - 1 - i) * (n - i)) / 2 + col] * raw_x_A[col];
//            }
//            raw_x_A[n - 1 - i] = round(
//                    (y_A[n - 1 - i] - sum) / R_sA[(n - 1 - i) * n - ((n - 1 - i) * (n - i)) / 2 + n - 1 - i]);
//            sum = 0;
//        }
//    }
//tiling
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        double sum = 0.0;
//
//        // store index in register
//        // Multiplication is not executed in every iteration.
//        //int idx_Ai = idx * n;
        for (int col = idx + 1; col < n; col++)
            sum += R_sA[idx * n + col] * x_A[col];
        x_A[idx] = round((y_A[idx] - sum) / R_sA[idx * n + idx]);


//        int idx_Ai = idx * n;
//        for (int j = 0; j < n; j++)
//            if (idx != j)
//                sum += R_sA[idx_Ai + j] * x_A[j];
//        x_Next_A[idx] = round((y_A[idx] - sum) / R_sA[idx * n + idx]);
    }

}

void testDevice(int devID) {
    // Check if we can run. Maybe do something more...
    hipDeviceProp_t deviceProp;

    hipGetDeviceProperties(&deviceProp, devID);
    if (deviceProp.major == 9999 && deviceProp.minor == 9999) {   /* Simulated device. */
        printf("There is no device supporting CUDA.\n");
        hipDeviceReset();
    } else
        printf("Using GPU device number %d.\n", devID);
}

double run(int n, int nswp, Babai_search_asyn bsa) {
    double *x, *x_A, *x_Next_A, *y_A, *R_sA;

    x = (double *) malloc(n * sizeof(double));

    hipMallocManaged(&x_A, n * sizeof(double));
    hipMallocManaged(&x_Next_A, n * sizeof(double));
    hipMallocManaged(&y_A, n * sizeof(double));
    hipMallocManaged(&R_sA, bsa.R_A.size() * sizeof(double));

    //x = bsa.x_A.data();
    x[n - 1] = round(bsa.y(n - 1) / bsa.R(n - 1, n - 1));
    for (int i = 0; i < n; i++) {
        y_A[i] = bsa.y_A[i];
    }
    for (int i = 0; i < bsa.R_A.size(); i++) {
        R_sA[i] = bsa.R_A[i];
    }
    x_A[n - 1] = x_Next_A[n - 1] = x[n - 1];

    hipMemcpy(y_A, bsa.y_A.data(), n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(R_sA, bsa.R_A.data(), bsa.R_A.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(x_A, x, n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(x_Next_A, x, n * sizeof(double), hipMemcpyHostToDevice);

    int tileSize = 4;
    // Optimized kernel
    int nTiles = n / tileSize + (n % tileSize == 0 ? 0 : 1);
    int gridHeight = n / tileSize + (n % tileSize == 0 ? 0 : 1);
    int gridWidth = n / tileSize + (n % tileSize == 0 ? 0 : 1);
    //printf("w=%d, h=%d\n", gridWidth, gridHeight);
    dim3 dGrid(gridHeight, gridWidth), dBlock(tileSize, tileSize);

    std::clock_t start = std::clock();
    for (int k = 0; k < nswp; k++) {
        //if (k % 2)
            find_raw_x0_cuda<<<nTiles, tileSize>>>(n, x_A, x_Next_A, y_A, R_sA);
        //else
            //find_raw_x0_cuda<<<nTiles, tileSize>>>(n, x_Next_A, x_A, y_A, R_sA);


    }

    hipDeviceSynchronize();
    double time = (std::clock() - start) / (double) CLOCKS_PER_SEC;

    double res = 0.0f;
    VectorXd x_result = VectorXd(n);

    for (int i = 0; i < n; i++) {
        x_result(i) = x_A[i];
    }

    res = (bsa.y - bsa.R * x_result).norm();

    printf("Sweep: %d, Res: %.5f, Run time: %fs\n", nswp, res, time);

    hipFree(x_A);
    hipFree(x_Next_A);
    hipFree(y_A);
    hipFree(R_sA);
    free(x);
    return res;
}

int main() {

    testDevice(0);

    int n = 2048, n_jobs = 50;
    Babai_search_asyn bsa(n);

    bsa.init(true, true, 0.1);

    std::cout << "find_raw_x0" << std::endl;
    bsa.search_vec(0);

//    std::cout << "find_raw_x0_OMP" << std::endl;
//    for(int nswp = 5; nswp <= n_jobs; nswp++)
//        for(int j = 10; j <= n_jobs; j++)
//            VectorXd x_par = bsa.find_raw_x0_OMP(j, nswp);

    std::vector<double> nswp_pl(20), res_pl(20), tim_pl(20);
    for (int nswp = 0; nswp < 20; nswp++) {
        nswp_pl.push_back(nswp);
        res_pl.push_back(bsa.init_res);
        tim_pl.push_back(bsa.search_vec(0));
    }

    const std::map<std::string, std::string> keyword_arg{
            {"marker",     "o"},
            {"markersize", "5"},
            {"label",      "Serial"}
    };

    plt::xlim(1, 20);
    plt::plot(nswp_pl, tim_pl, keyword_arg);

    const std::map<std::string, std::string> keyword_arg2{
            {"marker",     "1"},
            {"markersize", "5"},
            {"label",      "Matlab"}
    };

    string tim =
            "/home/shilei/CLionProjects/babai_asyn/data/Res_" + to_string(n) + ".csv";
    string row_string, entry;
    int index = 0;
    vector<double> nswp_pl2(20), tim_pl2(20);
    ifstream f1(tim);
    while (getline(f1, row_string)) {
        double d = stod(row_string);
        nswp_pl2.push_back(index);
        tim_pl2.push_back(d);
        index++;
    }

    plt::xlim(1, 20);
    plt::plot(nswp_pl2, tim_pl2, keyword_arg2);

    std::cout << "find_raw_x0_CUDA" << std::endl;
    vector<double> nswp_pl3(20), tim_pl3(20);
    for (int nswp = 10; nswp <= 200; nswp += 10) {
        double time = run(n, nswp, bsa);
        tim_pl3.push_back(time);
    }


    const std::map<std::string, std::string> keyword_arg3{
            {"marker",     "x"},
            {"markersize", "5"},
            {"label",      "GPU"}
    };

    plt::xlim(1, 20);
    plt::plot(nswp_pl, tim_pl3, keyword_arg3);

    plt::title("Residual with GPU");
    plt::legend();
    plt::xlabel("Num of iterations x 10");
    plt::ylabel("Residual");
    plt::save("./resCUDA.png");

    return 0;

}
